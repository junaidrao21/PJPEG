#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>

//#include "image.h"

#define TILE_WIDTH 16

//Code written for 1D arrays ahead of time
//for CUDA implementation. 
//
//Assumptions: the 'arr' has been malloc'd
//			   before the function call.
//
//Functionality: linearizes the traversal of
//			     the 8x8 block before the Huffman
//				 encoding scheme is used.
void Traverse(char *block, char *arr)
{
	int count = 0;
	int r = 0;
	int c = 0;

	while(count < 64)
	{
		//FIRST HALF OF TRAVERSE
		if(c < 7)
		{
			//MOVE RIGHT
			arr[count++] = block[r*8 + (c++)];
			
			//ALGORITHM ALWAYS ENDS HERE
			if(count == 64)
				break;
		}	
		else
		{
			//MOVE DOWN
			arr[count++] = block[(r++)*8 + c];
		}

		//MOVE DOWN AND LEFT	
		while((r<7) && (c>0))
		{
		    arr[count++] = block[(r++)*8 + (c--)];
		}



		//SECOND HALF OF TRAVERSE
		if(r < 7)
		{
			//MOVE DOWN
			arr[count++] = block[(r++)*8 + c];
		}
		else
		{
			//MOVE RIGHT
			arr[count++] = block[r*8 + (c++)];
		}

		//MOVE UP AND RIGHT	
		while((r>0) && (c<7))
		{
			arr[count++] = block[(r--)*8 + (c++)];
		}
	}
}



int main (int argc, char **argv)
{
//	int imageWidth, imageHeight, matSize;
//	int *hostInputImage, *hostOutputImage;
//	int *deviceInputImage, *deviceOutputImage;

	char test[64] = {  1,  2,  6,  7, 15, 16, 28, 29,
		          3,  5,  8, 14, 17, 27, 30, 43,
	                  4,  9, 13, 18, 26, 31, 42, 44,
			 10, 12, 19, 25, 32, 41, 45, 54,
			 11, 20, 24, 33, 40, 46, 53, 55,
			 21, 23, 34, 39, 47, 52, 56, 61,
	 		 22, 35, 38, 48, 51, 57, 60, 62,
			 36, 37, 49, 50, 58, 59, 63, 64};


	char *A = (char *)calloc(64, sizeof(char));
	char *B = (char *)calloc(64, sizeof(char));

	memcpy(A, test, 64*sizeof(char));

	Traverse(A, B);

	int i;
	for(i=0; i<64; i++)
		printf("B[%d] = (%d).\n", i, (int)B[i]);

    /*
	
	clock_t gpu_start, gpu_end;

	*if (argc != 5)
	{
		printf("Usage: ./lab2 <input-image> <output-image-name> <blur/gaussian/emboss/sharp> <1d/2d kernel>\n");
		exit(1);
	}*/

	// Read in image and convert to readable format
	//read_image_template<int>(argv[1], &hostInputImage, &imageWidth, &imageHeight);

	// Set image size information
	//int img_size = imageWidth * imageHeight * sizeof(int);

	// Allocate memory for image on GPU
	//cudaMalloc((void **)&deviceInputImage, img_size);
	//cudaMalloc((void **)&deviceOutputImage, img_size);
		
	// Copy image to device
	//cudaMemcpy( deviceInputImage, hostInputImage, img_size, cudaMemcpyHostToDevice );
	//cudaMemcpy( deviceMatrix, hostMatrix, sizeof(double)*matSize*matSize, cudaMemcpyHostToDevice );
	
	
	/*****Pre-Processing*****/
	
	
	
	/*****Transforming*******/
	
	
	/*****Quantization*******/
	
	
	/********Encoding********/
	
    
	return 0;
}
