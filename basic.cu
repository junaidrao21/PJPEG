#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "image.h"

#define TILE_WIDTH 16

//Code written for 1D arrays ahead of time
//for CUDA implementation. 
//
//Assumptions: the 'arr' has been malloc'd
//			   before the function call.
//
//Functionality: linearizes the traversal of
//			     the 8x8 block before the Huffman
//				 encoding scheme is used.
void Traverse(char *block, char *arr)
{
	int count = 0;
	int r = 0;
	int c = 0;

	while(count < 64)
	{
		//FIRST HALF OF TRAVERSE
		if(c < 7)
		{
			//MOVE RIGHT
			arr[count++] = block[r*8 + (c++)];
			
			//ALGORITHM ALWAYS ENDS HERE
			if(count == 64)
				break;
		}	
		else
		{
			//MOVE DOWN
			arr[count++] = block[(r++)*8 + c];
		}

		//MOVE DOWN AND LEFT	
		while((r>0) && (r<7) && (c>0) && (c<7))
		{
			arr[count++] = block[(r++)*8 + (c--)];
		}



		//SECOND HALF OF TRAVERSE
		if(r < 7)
		{
			//MOVE DOWN
			arr[count++] = block[(r--)*8 + c];
		}
		else
		{
			//MOVE RIGHT
			arr[count++] = block[r*8 + (c++)];
		}

		//MOVE UP AND RIGHT	
		while((r>0) && (r<7) && (c>0) && (c<7))
		{
			arr[count++] = block[(r--)*8 + (c++)];
		}
	}
}



int main (int argc, char **argv)
{
	int imageWidth, imageHeight, matSize;
	int *hostInputImage, *hostOutputImage;
	int *deviceInputImage, *deviceOutputImage;
	
	clock_t gpu_start, gpu_end;

	/*if (argc != 5)
	{
		printf("Usage: ./lab2 <input-image> <output-image-name> <blur/gaussian/emboss/sharp> <1d/2d kernel>\n");
		exit(1);
	}*/

	// Read in image and convert to readable format
	read_image_template<int>(argv[1], &hostInputImage, &imageWidth, &imageHeight);

	// Set image size information
	int img_size = imageWidth * imageHeight * sizeof(int);

	// Allocate memory for image on GPU
	//hipMalloc((void **)&deviceInputImage, img_size);
	//hipMalloc((void **)&deviceOutputImage, img_size);
		
	// Copy image to device
	//hipMemcpy( deviceInputImage, hostInputImage, img_size, hipMemcpyHostToDevice );
	//hipMemcpy( deviceMatrix, hostMatrix, sizeof(double)*matSize*matSize, hipMemcpyHostToDevice );
	
	
	/*****Pre-Processing*****/
	
	
	
	/*****Transforming*******/
	
	
	/*****Quantization*******/
	
	
	/********Encoding********/
	

	return 0;
}