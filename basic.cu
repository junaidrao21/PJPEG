#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "image.h"

#define TILE_WIDTH 16

int main (int argc, char **argv)
{
	int imageWidth, imageHeight, matSize;
	int *hostInputImage, *hostOutputImage;
	int *deviceInputImage, *deviceOutputImage;
	
	clock_t gpu_start, gpu_end;

	/*if (argc != 5)
	{
		printf("Usage: ./lab2 <input-image> <output-image-name> <blur/gaussian/emboss/sharp> <1d/2d kernel>\n");
		exit(1);
	}*/

	// Read in image and convert to readable format
	read_image_template<int>(argv[1], &hostInputImage, &imageWidth, &imageHeight);

	// Set image size information
	int img_size = imageWidth * imageHeight * sizeof(int);

	// Allocate memory for image on GPU
	//hipMalloc((void **)&deviceInputImage, img_size);
	//hipMalloc((void **)&deviceOutputImage, img_size);
		
	// Copy image to device
	//hipMemcpy( deviceInputImage, hostInputImage, img_size, hipMemcpyHostToDevice );
	//hipMemcpy( deviceMatrix, hostMatrix, sizeof(double)*matSize*matSize, hipMemcpyHostToDevice );
	
	
	/*****Pre-Processing*****/
	
	
	
	/*****Transforming*******/
	
	
	/*****Quantization*******/
	
	
	/********Encoding********/
	

	return 0;
}