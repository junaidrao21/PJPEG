#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define COEFFS(Cu,Cv,u,v) {\
			if (u==0) Cu = 1/sqrt(2); else Cu = 1.0;\
			if (v==0) Cv = 1/sqrt(2); else Cv = 1.0;}

//Function prototypes
void Traverse(char *block, char *arr, int row);
void Inverse(char *block, char *arr, int row);

__global__ void parallel_dct(int **ky, int **kcb, int **kcr, char **kdy,
			                        char **kdcb, char **kdcr, int row, int col)
{
	int idx = (threadIdx.x + blockIdx.x * blockDim.x)*8;
	int idy = (threadIdx.y + blockIdx.y * blockDim.y)*8;
	int i,j,x;
	if (idx < row && idy < col)
	{
			for(i = 0; i < 8; i++)
				for(j = 0; j < 8; j++)
				{
					temp_dcty[i*8+j] = 0;
					temp_dctcb[i*8+j] = 0;
					temp_dctcr[i*8+j] = 0;
					for(x = 0; x < 8; x++)
					{
						temp_dcty[i*8+j] += DCT[i][x] * (double)img_y[idx+x][idy+j];
						temp_dctcb[i*8+j] += DCT[i][x] * (double)img_cb[idx+x][idy+j];
						temp_dctcr[i*8+j] += DCT[i][x] * (double)img_cr[idx+x][idy+j];
					}
				}
			for(i = 0; i < 8; i++)
				for(j = 0; j < 8; j++)
				{
					img_dy[idx+i][idy+j] = 0;
					img_dcb[idx+i][idy+j] = 0;
					img_dcr[idx+i][idy+j] = 0;
					for(x = 0; x < 8; x++)
					{
						img_dy[idx+i][idy+j] += temp_dcty[i*8+x] * DCT[j][x];
						img_dcb[idx+i][idy+j] += temp_dctcb[i*8+x] * DCT[j][x];
						img_dcr[idx+i][idy+j] += temp_dctcr[i*8+x] * DCT[j][x];
					}
				}
	}
}


int main (int argc, char **argv)
{
	//Timing variables
	struct timeval startrgb, endrgb, startdct, enddct, startquant, endquant, starthuff, endhuff, startcmptot, endcmptot, startdectot, enddectot;
	gettimeofday(&startcmptot, NULL);

	//Declaration of variables
	FILE *f_in  = fopen(argv[1], "r");
	FILE *f_out = fopen("output.ppm", "w");
	char img_type[16];
	int row, col, char_val, orig_row, orig_col;
	int c, i, j, k, m, n, x, y;
	int counter = 0;
	double temp_y, temp_cb, temp_cr;
	double Ci, Cj;

	//Quantization Matrix
	unsigned char Q[8][8] = {	16,  11,  10,  16,  24,  40,  51,  61,
	 													12,  12,  14,  19,  26,  58,  60,  55,
														14,  13,  16,  24,  40,  57,  69,  56,
														14,  17,  22,  29,  51,  87,  80,  62,
														18,  22,  37,  56,  68, 109, 103,  77,
														24,  35,  55,  64,  81, 104, 113,  92,
														49,  64,  78,  87, 103, 121, 120, 101,
														72,  92,  95,  98, 112, 100, 103,  99};

	//DCT cosine matrix
	// double DCT[8][8] =  {       sqrt(2)/4,         sqrt(2)/4,         sqrt(2)/4,         sqrt(2)/4,         sqrt(2)/4,         sqrt(2)/4,         sqrt(2)/4,          sqrt(2)/4,
	// 	    	         		   cos(M_PI/16)/2,  cos(3*M_PI/16)/2,  cos(5*M_PI/16)/2,  cos(7*M_PI/16)/2,  cos(9*M_PI/16)/2, cos(11*M_PI/16)/2, cos(13*M_PI/16)/2,  cos(15*M_PI/16)/2,
	//                      cos(2*M_PI/16)/2,  cos(6*M_PI/16)/2, cos(10*M_PI/16)/2, cos(14*M_PI/16)/2, cos(18*M_PI/16)/2, cos(22*M_PI/16)/2, cos(26*M_PI/16)/2,  cos(30*M_PI/16)/2,
	//                      cos(3*M_PI/16)/2,  cos(9*M_PI/16)/2, cos(15*M_PI/16)/2, cos(21*M_PI/16)/2, cos(27*M_PI/16)/2, cos(33*M_PI/16)/2, cos(39*M_PI/16)/2,  cos(45*M_PI/16)/2,
	//                      cos(4*M_PI/16)/2, cos(12*M_PI/16)/2, cos(20*M_PI/16)/2, cos(28*M_PI/16)/2, cos(36*M_PI/16)/2, cos(44*M_PI/16)/2, cos(52*M_PI/16)/2,  cos(60*M_PI/16)/2,
	//                      cos(5*M_PI/16)/2, cos(15*M_PI/16)/2, cos(25*M_PI/16)/2, cos(35*M_PI/16)/2, cos(45*M_PI/16)/2, cos(55*M_PI/16)/2, cos(65*M_PI/16)/2,  cos(75*M_PI/16)/2,
	//                      cos(6*M_PI/16)/2, cos(18*M_PI/16)/2, cos(30*M_PI/16)/2, cos(42*M_PI/16)/2, cos(54*M_PI/16)/2, cos(66*M_PI/16)/2, cos(78*M_PI/16)/2,  cos(90*M_PI/16)/2,
	//                      cos(7*M_PI/16)/2, cos(21*M_PI/16)/2, cos(35*M_PI/16)/2, cos(49*M_PI/16)/2, cos(63*M_PI/16)/2, cos(77*M_PI/16)/2, cos(91*M_PI/16)/2, cos(105*M_PI/16)/2};

	double DCT[8][8] =  {0.3536,  0.3536,  0.3536,  0.3536,  0.3536,  0.3536,  0.3536,  0.3536,
											 0.4904,  0.4157,  0.2778,  0.0975, -0.0975, -0.2778, -0.4157, -0.4904,
											 0.4619,  0.1913, -0.1913, -0.4619, -0.4619, -0.1913,  0.1913,  0.4619,
											 0.4157, -0.0975, -0.4904, -0.2778,  0.2778,  0.4904,  0.0975, -0.4157,
											 0.3536, -0.3536, -0.3536,  0.3536,  0.3536, -0.3536, -0.3536,  0.3536,
											 0.2778, -0.4904,  0.0975,  0.4157, -0.4157, -0.0975,  0.4904, -0.2778,
											 0.1913, -0.4619,  0.4619, -0.1913, -0.1913,  0.4619, -0.4619,  0.1913,
											 0.0975, -0.2778,  0.4157, -0.4904,  0.4904, -0.4157,  0.2788, -0.0975};

	double IDCT[8][8] = {0.3535,  0.4905,  0.4620,  0.4157,  0.3535,  0.2778,  0.1914,  0.0975,
											 0.3536,  0.4156,  0.1914, -0.0975, -0.3536, -0.4903, -0.4621, -0.2777,
											 0.3534,  0.2780, -0.1914, -0.4905, -0.3534,  0.0973,  0.4622,  0.4156,
											 0.3537,  0.0973, -0.4619, -0.2778,  0.3533,  0.4160, -0.1916, -0.4903,
											 0.3533, -0.0973, -0.4621,  0.2778,  0.3537, -0.4160, -0.1911,  0.4903,
											 0.3537, -0.2780, -0.1913,  0.4905, -0.3537, -0.0973,  0.4618, -0.4156,
											 0.3534, -0.4156,  0.1913,  0.0975, -0.3534,  0.4903, -0.4619,  0.2777,
											 0.3535, -0.4905,  0.4620, -0.4157,  0.3535, -0.2778,  0.1913, -0.0975};

	//Parse header
	fscanf(f_in, "%s\n", img_type);
	fscanf(f_in, "%d %d\n", &orig_col, &orig_row);
	fscanf(f_in, "%d\n", &char_val);

	//Pad row and col if necessary
	row = orig_row + (orig_row % 8);
	col = orig_col + (orig_col % 8);

	//Full RGB matrix
	unsigned char **img_c = (unsigned char **)calloc(row, sizeof(char *));

	//Separate YCbCr matrices
	int **img_y  = (int **)calloc(row, sizeof(int *));
	int **img_cb = (int **)calloc(row, sizeof(int *));
	int **img_cr = (int **)calloc(row, sizeof(int *));

	//Discrete cosine transform matrices
	double **img_dy  = (double **)calloc(row, sizeof(double *));
	double **img_dcb = (double **)calloc(row, sizeof(double *));
	double **img_dcr = (double **)calloc(row, sizeof(double *));

	//Quantization matrices (1D)
	char *img_qy  = (char *)calloc(row*col, sizeof(char));
	char *img_qcb = (char *)calloc(row*col, sizeof(char));
	char *img_qcr = (char *)calloc(row*col, sizeof(char));

	//Temp arrays for Traverse()
	char *trav_arr_qy  = (char *)calloc(64, sizeof(char));
	char *trav_arr_qcb = (char *)calloc(64, sizeof(char));
	char *trav_arr_qcr = (char *)calloc(64, sizeof(char));

	//Rearranged matrix for huffman (1D)
	char *huff = (char *)calloc(row*col*3, sizeof(char));

	//Allocate 2D arrays
	for(i=0; i<col; i++)
	{
		img_c[i]   = (unsigned char *)calloc(col*3, sizeof(char));
	  img_y[i]   = (int *)calloc(col, sizeof(int));
		img_cb[i]  = (int *)calloc(col, sizeof(int));
		img_cr[i]  = (int *)calloc(col, sizeof(int));
		img_dy[i]  = (double *)calloc(col, sizeof(double));
		img_dcb[i] = (double *)calloc(col, sizeof(double));
		img_dcr[i] = (double *)calloc(col, sizeof(double));
	}

	//Read in pixel data
	for(i=0; i<row; i++)
		for(j=0; j<col*3; j++)
			fscanf(f_in, "%c", &img_c[i][j]);

	// printf("RGB[0][0]\n");
	// printf("%8d %8d %8d\n",img_c[0][0], img_c[0][1], img_c[0][2]);

	//RBG -> YCbCr
	gettimeofday(&startrgb, NULL);
	for(i=0; i<row; i++)
		for(j=0,k=0; j<col*3; j+=3,k++)
		{
			img_y[i][k]  = (0.299)*img_c[i][j] + (0.587)*img_c[i][j+1] + (0.114)*img_c[i][j+2];
		  img_cb[i][k] = 128 - (0.168736)*img_c[i][j] - (0.331264)*img_c[i][j+1] + (0.5)*img_c[i][j+2];
		  img_cr[i][k] = 128 + (0.5)*img_c[i][j] - (0.418688)*img_c[i][j+1] - (0.081312)*img_c[i][j+2];
		}

	// printf("YCbCr[0][0]\n");
	// printf("%8d %8d %8d\n",img_y[0][0], img_cb[0][0], img_cr[0][0]);

	//Center
	for(i=0; i<row; i++)		// for the padded 0's does this mess anything up?
		for(j=0; j<col; j++)
		{
			img_y[i][j]  -= 128;
			img_cb[i][j] -= 128;
			img_cr[i][j] -= 128;
		}
	gettimeofday(&endrgb, NULL);

	printf("Before DCT");
	for(m=0;m<8;m++){
		for(n=0;n<8;n++)
			printf("%4d", img_cr[m][n]);
		printf("\n");
	}

	//Discrete Cosine Transform
	gettimeofday(&startdct, NULL);
	for(m=0; m<row; m+=8)
		for(n=0; n<col; n+=8)
			for(i = 0; i < 8; i++)
				for(j = 0; j < 8; j++)
				{
					temp_y  = 0.0;
					temp_cb = 0.0;
					temp_cr = 0.0;
				  for (x=0; x<8; x++)
				  	for (y=0; y<8; y++)
						{
							// temp_y  += DCT[x][i] * DCT[y][j] * img_y[m+x][n+y];
							// temp_cb += DCT[x][i] * DCT[y][j] * img_cb[m+x][n+y];
							// temp_cr += DCT[x][i] * DCT[y][j] * img_cr[m+x][y+n];
							temp_y  += cos((double)(2*x+1)*(double)i*M_PI/16) * cos((double)(2*y+1)*(double)j*M_PI/16) * (double)img_y[m+x][n+y];
							temp_cb += cos((double)(2*x+1)*(double)i*M_PI/16) * cos((double)(2*y+1)*(double)j*M_PI/16) * (double)img_cb[m+x][n+y];
							temp_cr += cos((double)(2*x+1)*(double)i*M_PI/16) * cos((double)(2*y+1)*(double)j*M_PI/16) * (double)img_cr[m+x][n+y];
		 				}
					COEFFS(Ci,Cj,i,j);
					temp_y  *= Ci * Cj * 0.25;
					temp_cb *= Ci * Cj * 0.25;
					temp_cr *= Ci * Cj * 0.25;

				  img_dy[m+i][n+j]  = temp_y;
					img_dcb[m+i][n+j] = temp_cb;
					img_dcr[m+i][n+j] = temp_cr;
				}
	gettimeofday(&enddct, NULL);

	printf("After DCT\n");
	for(m=0;m<8;m++){
		for(n=0;n<8;n++)
			printf("%6.2f", img_dcr[m][n]);
		printf("\n");
	}

	//Quantization
	gettimeofday(&startquant, NULL);
	for(m=0; m<row; m+=8)
		for(n=0; n<col; n+=8)
			for(i=0; i<8; i++)
				for(j=0; j<8; j++)
				{
					img_qy[(m+i)*row + (n+j)]  = (char)rint((img_dy[m+i][n+j]/Q[i][j]));
					img_qcb[(m+i)*row + (n+j)] = (char)rint((img_dcb[m+i][n+j]/Q[i][j]));
					img_qcr[(m+i)*row + (n+j)] = (char)rint((img_dcr[m+i][n+j]/Q[i][j]));
				}
	gettimeofday(&endquant, NULL);

	// printf("After Quant\n");
	// for(m=0;m<8;m++){
	// 	for(n=0;n<8;n++)
	// 		printf("%6d", img_qcr[m*row+n]);
	// 	printf("\n");
	// }

	//Linearization of each 8x8 block before compression
	for(m=0; m<row; m+=8)
		for(n=0; n<col; n+=8)
		{
			//Linearization
			Traverse(img_qy+(m*row+n), trav_arr_qy, row);
			Traverse(img_qcb+(m*row+n), trav_arr_qcb, row);
			Traverse(img_qcr+(m*row+n), trav_arr_qcr, row);

			//Combination into single Huffman array
			for(c=0; c<64; c++, counter++)
			{
				huff[counter] = trav_arr_qy[c];
				huff[col*row+counter] = trav_arr_qcb[c];
				huff[col*row*2+counter] = trav_arr_qcr[c];
			}
		}

	// printf("After Traverse");
	// for(m=row*col*2;m<row*col*2+64;m++){
	// 	if(m%8 == 0)
	// 		printf("\n");
	// 	printf("%6d", huff[m]);
	// }
	// printf("\n");

	//Write out combined matrix to "output.ppm"
	fprintf(f_out, "%s\n", img_type);
	fprintf(f_out, "%d %d\n", col, row);
	fprintf(f_out, "%d\n", char_val);
	for(m=0; m<row*col*3; m++)
		fprintf(f_out, "%c", huff[m]);

	fclose(f_in);
	fclose(f_out);

	//Huffman Compression
	gettimeofday(&starthuff, NULL);
	// char arg_comp[100] = {"./huff -c output.ppm output.ppm.huf"};
	system("./huff -c output.ppm output.ppm.huf");
	gettimeofday(&endhuff, NULL);

	gettimeofday(&endcmptot, NULL);

	//Timing calculations
	double delta_us_rgb   = (double)(endrgb.tv_usec - startrgb.tv_usec) / 1000000 + (endrgb.tv_sec - startrgb.tv_sec);
	double delta_us_dct   = (double)(enddct.tv_usec - startdct.tv_usec) / 1000000 + (enddct.tv_sec - startdct.tv_sec);
	double delta_us_quant = (double)(endquant.tv_usec - startquant.tv_usec) / 1000000 + (endquant.tv_sec - startquant.tv_sec);
	double delta_us_huff  = (double)(endhuff.tv_usec - starthuff.tv_usec) / 1000000 + (endhuff.tv_sec - starthuff.tv_sec);
	double delta_us_cmptot   = (double)(endcmptot.tv_usec - startcmptot.tv_usec) / 1000000 + (endcmptot.tv_sec - startcmptot.tv_sec);

	//Timing outputs in milliseconds
	printf("RGB->YCbCr = %6.3f\n", delta_us_rgb);
	printf("DCT =        %6.3f\n", delta_us_dct);
	printf("Quant =      %6.3f\n", delta_us_quant);
	printf("Huffman =    %6.3f\n", delta_us_huff);
	printf("Compr Tot =  %6.3f\n", delta_us_cmptot);



	gettimeofday(&startdectot, NULL);
	//printf("Decompressing...\n");
	//Huffman Decompression
	// char arg_decomp[100] = {"./huff -d output.ppm.huf output.ppm.uhuf"};
	system("./huff -d output.ppm.huf output.ppm.uhuf");

	//Check to ensure Huffman was successful
	// char dif[100] = {"diff output.ppm output.ppm.uhuf"};
	system("diff output.ppm output.ppm.uhuf");

	//End of JPEG encoding



	//////////////////////////////////////////////////////////////////////////////
	//  BEGIN DECODING OF UNCOMPRESSED JPEG IMAGE
	//////////////////////////////////////////////////////////////////////////////

	//Open uncompressed huffman file
	FILE *g_in  = fopen("output.ppm.uhuf", "r");
	FILE *g_out = fopen("result.ppm", "w");

	//Parse header
	fscanf(g_in, "%s\n", img_type);
	fscanf(g_in, "%d %d\n", &col, &row);
	fscanf(g_in, "%d\n", &char_val);


	//Prepare huffman array
	memset(huff, 1, row*col*3*sizeof(char));

	//printf("Reading in file\n");
	//Read in file contents
	for(m=0; m<row*col*3; m++)
			fscanf(g_in, "%c", &huff[m]);
	printf("m = (%d).\n", m);

	// printf("Before ITraverse");
	// for(m=row*col*2;m<row*col*2+64;m++){
	// 	if(m%8 == 0)
	// 		printf("\n");
	// 	printf("%6d", huff[m]);
	// }
	// printf("\n");

	//printf("Reverse Linear\n");
	//Reverse the linearization
	counter = 0;
	for(i=0; i<3; i++)
		for(m=0; m<row; m+=8)
			for(n=0; n<col; n+=8, counter++)
				if(i==0)
					Inverse(&img_qy[m * row + n], (huff + counter*64), row);
				else if(i==1)
					Inverse(&img_qcb[m * row + n], (huff + counter*64), row);
				else
					Inverse(&img_qcr[m * row + n], (huff + counter*64), row);

	// printf("Before IQuant\n");
	// for(m=0;m<8;m++){
	// 	for(n=0;n<8;n++)
	// 		printf("%6d", img_qcr[m*row+n]);
	// 	printf("\n");
	// }

	//printf("IQuant\n");
	//Inverse Quantization
	for(m=0; m<row; m+=8)
		for(n=0; n<col; n+=8)
			for (i=0; i<8; i++)
				for (j=0; j<8; j++)
				{
							img_dy[m+i][n+j] = (img_qy[(m+i)*row + (n+j)]*Q[i][j]);
							img_dcb[m+i][n+j] = (img_qcb[(m+i)*row + (n+j)]*Q[i][j]);
							img_dcr[m+i][n+j] = (img_qcr[(m+i)*row + (n+j)]*Q[i][j]);
				}

	printf("Before IDCT\n");
	for(m=0;m<8;m++){
		for(n=0;n<8;n++)
			printf("%6.2f", img_dcr[m][n]);
		printf("\n");
	}

	//printf("IDCT\n");
	//Inverse Discrete Cosine Transform
	for(m=0; m<row; m+=8)
		for(n=0; n<col; n+=8)
			for(x=0; x<8; x++)
				for(y=0; y<8; y++)
				{
					temp_y = 0.0;
					temp_cb = 0.0;
					temp_cr = 0.0;
				  for (i=0; i<8; i++)
				  	for (j=0; j<8; j++)
						{
							COEFFS(Ci,Cj,i,j);
							// temp_y  += Ci * Cj * DCT[x][i] * DCT[y][j] * img_dy[m+i][n+j];
							// temp_cb += Ci * Cj * DCT[x][i] * DCT[y][j] * img_dcb[m+i][n+j];
							// temp_cr += Ci * Cj * DCT[x][i] * DCT[y][j] * img_dcr[m+i][n+j];
							temp_y  += Ci * Cj * cos((double)(2*x+1)*(double)i*M_PI/16) * cos((double)(2*y+1)*(double)j*M_PI/16) * (double)img_dy[m+i][n+j];
							temp_cb += Ci * Cj * cos((double)(2*x+1)*(double)i*M_PI/16) * cos((double)(2*y+1)*(double)j*M_PI/16) * (double)img_dcb[m+i][n+j];
							temp_cr += Ci * Cj * cos((double)(2*x+1)*(double)i*M_PI/16) * cos((double)(2*y+1)*(double)j*M_PI/16) * (double)img_dcr[m+i][n+j];
		 				}
					temp_y  /= 4;
					temp_cb /= 4;
					temp_cr /= 4;

				  img_y[m+x][n+y]  = rint(temp_y);
					img_cb[m+x][n+y] = rint(temp_cb);
					img_cr[m+x][n+y] = rint(temp_cr);
				}

	printf("After IDCT\n");
	for(m=0;m<8;m++){
		for(n=0;n<8;n++)
			printf("%6d", img_cr[m][n]);
		printf("\n");
	}

	//Un-Center
	for(i=0; i<row; i++)
		for(j=0; j<col; j++)
		{
			img_y[i][j]  += 128;
			img_cb[i][j] += 128;
			img_cr[i][j] += 128;
		}
		// printf("YCbCr[0][0]\n");
		// printf("%8d %8d %8d\n",img_y[0][0], img_cb[0][0], img_cr[0][0]);


	//printf("YCbCr->RGB\n");
	//YCbCr back to RGB
	for(m=0; m<row; m++)
		for(n=0, j=0; n<col; n++, j+=3)
		{
			img_c[m][j]   = (unsigned char)(img_y[m][n] + 1.40200 * (img_cr[m][n] - 128));
			img_c[m][j+1] = (unsigned char)(img_y[m][n] - 0.34414 * (img_cb[m][n] - 128) - 0.71414 * (img_cr[m][n] - 128));
			img_c[m][j+2] = (unsigned char)(img_y[m][n] + 1.77200 * (img_cb[m][n] - 128));
		}

		// printf("RGB[0][0]\n");
		// printf("%8d %8d %8d\n",img_c[0][0], img_c[0][1], img_c[0][2]);

	//printf("Write out\n");
	//Write out the final, reconstructed RGB image
	fprintf(g_out, "%s\n", img_type);
	fprintf(g_out, "%d %d\n", col, row);
	fprintf(g_out, "%d\n", char_val);
	for(m=0; m<orig_row; m++)
		for(n=0; n<orig_col*3; n++)
			fprintf(g_out, "%c", img_c[m][n]);

	//Clean up
	fclose(g_in);
	fclose(g_out);
	//printf("freeing 2d memory\n");
	//Free allocated memory
	for(i=0; i<row; i++)
	{
		free(img_c[i]);
		free(img_y[i]);
		free(img_cb[i]);
		free(img_cr[i]);
		free(img_dy[i]);
		free(img_dcb[i]);
		free(img_dcr[i]);
	}
	// for(i=0;i<orig_row;i++)
	// 	free(img_final[i]);
	free(img_c);
	free(img_y);
	free(img_cb);
	free(img_cr);
	free(img_dy);
	free(img_dcb);
	free(img_qy);
	free(img_qcb);
	free(img_qcr);
	free(trav_arr_qy);
	free(trav_arr_qcb);
	free(trav_arr_qcr);
	free(huff);
	//free(img_final);

	// char remove_excess_ppms[100] = {"rm output.ppm output.ppm.huf output.ppm.uhuf"};
	// system(remove_excess_ppms);
	gettimeofday(&enddectot, NULL);
	double delta_us_dectot   = (double)(enddectot.tv_usec - startdectot.tv_usec) / 1000000 + (enddectot.tv_sec - startdectot.tv_sec);
	printf("Decom Tot =  %6.3f\n", delta_us_dectot);

	//Scott is n00b
	return 0;
}

//////////////////////////////////////////////////////////////////////////////
//  FUNCTION DEFINITIONS of Traverse() and Inverse()
//////////////////////////////////////////////////////////////////////////////
void Traverse(char *block, char *arr, int row)
{
	int count = 0;
	int r = 0;
	int c = 0;

	while(count < 64)
	{
		if(c < 7)
		{
			arr[count++] = block[(r*row) + (c++)];
			if(count == 64)
				break;
		}
		else
			arr[count++] = block[(r++)*row + c];

		while((r<7) && (c>0))
		    arr[count++] = block[(r++)*row + (c--)];

		if(r < 7)
			arr[count++] = block[(r++)*row + c];
		else
			arr[count++] = block[(r*row) + (c++)];

		while((r>0) && (c<7))
			arr[count++] = block[(r--)*row + (c++)];
	}
}

void Inverse(char *block, char *arr, int row)
{
	int count = 0;
	int r = 0;
	int c = 0;

	while(count < 64)
	{
		if(c < 7)
		{
			block[(r*row) + (c++)] = arr[count++];

			if(count == 64)
				break;
		}
		else
			block[(r++)*row + c] = arr[count++];

		while((r<7) && (c>0))
		    block[(r++)*row + (c--)] = arr[count++];

		if(r < 7)
			block[(r++)*row + c] = arr[count++];
		else
			block[(r*row) + (c++)] = arr[count++];

		while((r>0) && (c<7))
			block[(r--)*row + (c++)] = arr[count++];
	}
}
